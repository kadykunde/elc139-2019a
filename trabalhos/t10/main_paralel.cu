#include "hip/hip_runtime.h"



#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/time.h>

#include "Cube_Unity.h"
#include "bmp.h"

#define PI 3.1415

__global__ void render(Cube_Unity *Cube, Cube_Unity *Cube_Perspective){
	int Size = 100;
	int width = 2000;
	int OffSet = 3000;
   double startAngulo = 1;
   double offset = startAngulo;
   
   auto k = blockIdx.x;
   int i = int(threadIdx.x/10);
   
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
            Cube[i + Size * (j + Size * k)].z += offset;
      }
   
	double angulo = startAngulo * PI/180;

   double matrix_x[4][4] = { cos(angulo), 0, sin(angulo), 0,
                                      0, 1,           0, 0,
                           -sin(angulo), 0, cos(angulo), 0,
                                      0, 0,           0, 1};

   double new_x, new_y, new_z;

      for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
            new_x = Cube[i + Size * (j + Size * k)].x * matrix_x[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_x[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_x[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_x[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_x[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;
      }
   
	angulo = startAngulo * PI/180;

   double matrix_y[4][4] = {1,           0,            0, 0,
                           0, cos(angulo), -sin(angulo), 0,
                           0, sin(angulo),  cos(angulo), 0,
                           0,           0,            0, 1};

						   
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){

            new_x = Cube[i + Size * (j + Size * k)].x * matrix_y[0][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[0][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[0][2];
            new_y = Cube[i + Size * (j + Size * k)].x * matrix_y[1][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[1][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[1][2];
            new_z = Cube[i + Size * (j + Size * k)].x * matrix_y[2][0] + Cube[i + Size * (j + Size * k)].y * matrix_y[2][1] + Cube[i + Size * (j + Size * k)].z * matrix_y[2][2];

            Cube[i + Size * (j + Size * k)].x = new_x;
            Cube[i + Size * (j + Size * k)].y = new_y;
            Cube[i + Size * (j + Size * k)].z = new_z;
      }
   offset = OffSet;
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
            Cube[i + Size * (j + Size * k)].z += offset;
      }
   
   double dist_to_screen = 300;

	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
            new_x = Cube[i + Size * (j + Size * k)].x*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
            new_y = Cube[i + Size * (j + Size * k)].y*dist_to_screen/Cube[i + Size * (j + Size * k)].z+width/2;
			Cube_Perspective[i + Size * (j + Size * k)].x = new_x;
			Cube_Perspective[i + Size * (j + Size * k)].y = new_y;
			Cube_Perspective[i + Size * (j + Size * k)].z = 0;
			Cube_Perspective[i + Size * (j + Size * k)].cor = Cube[i + Size * (j + Size * k)].cor;
      }
   offset = -OffSet;
	  for(auto j = (threadIdx.x*10)%100; j < ((threadIdx.x*10)%100)+10; j++){
            Cube[i + Size * (j + Size * k)].z += offset;
      }
}


int main(void)
{

	

	Cube_Unity *Cube;
	Cube_Unity *Cube_Perspective;
	
	int Size = 100;
	int width = 2000;
	int frames = 1;
	double spacing_factor = 40;

   double inix, iniy, iniz;
   double atualx, atualy, atualz;
   double spacing;

   inix = iniy = iniz = atualx = atualy = atualz = -double(Size)/2 * spacing_factor;
   spacing = -inix*2 / double(Size);

   hipMallocManaged(&Cube, Size*Size*Size*sizeof(Cube_Unity));
   hipMallocManaged(&Cube_Perspective, Size*Size*Size*sizeof(Cube_Unity));
   for(auto i = 0; i < Size; i++){
       for(auto j = 0; j < Size; j++){
           for(auto k = 0; k < Size; k++){
              Cube[i + Size * (j + Size * k)].set_pos(atualx,atualy,atualz, 255);
              atualz += spacing;
           }
           atualz = iniz;
           atualy += spacing;
       }
       atualy = iniy;
       atualx += spacing;
   }

   // allocate picture array
   unsigned char* pic = new unsigned char[frames * width * width];

   for (int frame = 0; frame < frames; frame++) {
      for (int row = 0; row < width; row++) {
         for (int col = 0; col < width; col++) {
            unsigned char color = (unsigned char) 255;
            pic[frame * width * width + row * width + col] = (unsigned char) color;
         }
      }
   }

	int max_threads, max_blocks;

	max_threads = 1000;
	max_blocks = Size;
	
// start time
	timeval start, end;
	gettimeofday(&start, NULL);
	
   for(int frame = 0; frame < frames; frame++){
	  render<<<max_blocks, max_threads>>>(Cube, Cube_Perspective);
      //render();
	  hipDeviceSynchronize();
      for(auto i = 0; i < Size; i++){
         for(auto j = 0; j < Size; j++){
            for(auto k = 0; k < Size; k++){
               int row = (int)Cube_Perspective[i + Size * (j + Size * k)].x;
               int col = (int)Cube_Perspective[i + Size * (j + Size * k)].y;
               if(row >= 0 && row < width && col >= 0 && col < width){
                  unsigned char color = (unsigned char) 0;
                  pic[frame * width * width + row * width + col] = (unsigned char) color;
               }
            }
         }
      }
   }

	gettimeofday(&end, NULL);
	double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
	std::cout << "compute time: " << runtime << " s\n";

    for (int frame = 0; frame < frames; frame++) {
      char name[32];
      sprintf(name, "cube%d.bmp", frame + 1000);
      writeBMP(width, width, &pic[frame * width * width], name);
    }


}
